#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define NUM_ALPHABETS 26
#define MAX_DG        10
#define PWD_LEN       3

__device__ const unsigned int s_table[] = 
{
    7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22 ,
    5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20 ,
    4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23 ,
    6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21 
};

__device__ const unsigned int k_table[] = 
{
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee ,
    0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501 ,
    0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be ,
    0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821 ,
    0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa ,
    0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8 ,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed ,
    0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a ,
    0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c ,
    0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70 ,
    0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05 ,
    0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665 ,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039 ,
    0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1 ,
    0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1 ,
    0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 
};

__device__ const int digests_3letters[] = 
{
    0xbc519d9f, 0xca21ef70, 0x07f3145c, 0xd8290a98, 
    0xe9475db1, 0x63ee3198, 0xf37cf4e3, 0x9a8e47d4, 
    0x76dab734, 0x98d2214b, 0x047d30ef, 0xc52d15d8, 
    0x1c77e47f, 0xeb228a00, 0x7df43d76, 0xaac6e219, 
    0x06a3c1b8, 0x7e246791, 0xdaf00335, 0x23576cba, 
    0x2285c2d4, 0x1d539374, 0x0a147705, 0x6439d01e, 
    0x4b90f674, 0x5fdedeb8, 0xc2dcad8e, 0xce312a0a, 
    0xb2211727, 0x6962b154, 0xf5c3146e, 0xe3f86d5a, 
    0xb860bfb3, 0xb2aeeb51, 0xa3018b76, 0x2ff32e2e, 
    0x747d75a4, 0x483bff19, 0x59902ee9, 0x48750e6f 
};

__device__ void md5(char* message, int length, unsigned int* digest) 
{
	// Encryption init values (key)
   	unsigned int a0 = 0x67452301;
	unsigned int b0 = 0xefcdab89; 
   	unsigned int c0 = 0x98badcfe; 
   	unsigned int d0 = 0x10325476; 

	// Init values
	unsigned int A = a0;
	unsigned int B = b0;
	unsigned int C = c0;
	unsigned int D = d0;

	// Encrypted message
	unsigned int M[16] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0};

	// Move the input password to the temp buffer message
	memcpy(M,message,length);
	// Attach a termination character at the end of the input password inside the temp buffer
	((char*)M)[length]=0x80;
	// Add the length * 8 value in last but one position of the array
	M[14]=length*8;
	// Loop 64 times
	for (int i = 0;i < 64;i++) 
	{
		// Encryption logic
		unsigned int F = (B & C) | ((~B) & D);
		unsigned int G = (D & B) | ((~D) & C);
		unsigned int H = B ^ C ^ D;
		unsigned int I = C ^ (B | (~D));
		unsigned int tempD = D;
		D = C;
		C = B;
		unsigned int X = I;
		unsigned int g = (7*i) & 15;
		if (i < 48) { X = H; g = (3*i+5) & 15; }
		if (i < 32) { X = G; g = (5*i+1) & 15; }
		if (i < 16) { X = F; g = i; }

		unsigned int tmp = A + X + k_table[i] + M[g];
		B = B + ((tmp << s_table[i]) | ((tmp & 0xffffffff) >> (32-s_table[i])));
		A = tempD;
	}
	// Store the encrypted password
	digest[0] = a0 + A;
	digest[1] = b0 + B;
	digest[2] = c0 + C;
	digest[3] = d0 + D;
}

__device__ int pwd_num = 0;

__global__ void passwordCrackKernel(char* match_d)
{
    unsigned int dg[4];
    char pwd[PWD_LEN + 1];
    // Get the thread and block ids for iteration
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k = threadIdx.y;
    // Index validity check
    if (i + j + k <= PWD_LEN * (NUM_ALPHABETS - 1))
    {
        // Initialize the password with a character sequence for comparison
        pwd[0] = 'a' + i;
        pwd[1] = 'a' + j;
        pwd[2] = 'a' + k;
        pwd[3] = 0;

        // Get the encrypted version of the input password sequence
        md5(pwd, PWD_LEN, dg);
        // Loop through all the digest combination for a match
        for (int ii = 0;ii < MAX_DG ;ii++)
        {
            // Check for a encryption hit
            if (( dg[0] == digests_3letters[ii*4] ) && ( dg[1] == digests_3letters[ii*4+1] ) && ( dg[2] == digests_3letters[ii*4+2] ) && ( dg[3] == digests_3letters[ii*4+3] )) 
            {
                // Use a temporary pointer to point to next available empty location
                char* temp = &match_d[pwd_num];
                // Copy the matching password to the location specified
                memcpy(temp,pwd,PWD_LEN);
                // Increement the password indicator to the next location
                pwd_num += PWD_LEN;
            }
        }
    }
}

// Main function
int main()
{
    // Local variables for host and device
    char match_h[PWD_LEN*MAX_DG];
    char* match_d;

    // Allocate the memory for the pointers
    hipMalloc((void**) &match_d, PWD_LEN*MAX_DG);

    // CUDA kernel block and grid dimensions
    dim3 threadsPerBlock(NUM_ALPHABETS, NUM_ALPHABETS);
    dim3 blocksPerGrid(NUM_ALPHABETS);

	clock_t begin = clock();

    // Invoke cuda kernel
    passwordCrackKernel<<<blocksPerGrid, threadsPerBlock>>>(match_d);

    // Synchronize all the threads
    hipDeviceSynchronize();

    // Copy the result from device to host
    hipMemcpy(match_h, match_d, PWD_LEN*MAX_DG, hipMemcpyDeviceToHost);

    // Print the password
    for (int i = 0; i < PWD_LEN*MAX_DG; i = i+PWD_LEN)
    {
        printf("%c", match_h[i]);
        printf("%c", match_h[i+1]);
        printf("%c\n", match_h[i+2]);
    }

	printf("\nElapsed time: %f seconds\n", (double)(clock() - begin) / CLOCKS_PER_SEC);

    // Free the pointers
    hipFree(match_d);
}