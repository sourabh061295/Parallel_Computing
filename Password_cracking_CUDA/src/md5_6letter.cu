#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define NUM_ALPHABETS 26
#define MAX_DG        10
#define PWD_LEN       6

__device__ const unsigned int s_table[] = 
{
    7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22 ,
    5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20 ,
    4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23 ,
    6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21 
};

__device__ const unsigned int k_table[] = 
{
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee ,
    0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501 ,
    0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be ,
    0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821 ,
    0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa ,
    0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8 ,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed ,
    0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a ,
    0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c ,
    0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70 ,
    0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05 ,
    0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665 ,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039 ,
    0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1 ,
    0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1 ,
    0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 
};

__device__ const int digests_6letters[] = 
{
    0xb15d1993, 0x8f7873ad, 0x509361e9, 0x87887d3a, 
    0x9ef17384, 0x21e22965, 0x759e4557, 0x85f4cbb4, 
    0x8770cff1, 0x1ebae88f, 0x0e30356d, 0x5bc404db, 
    0xe6679a67, 0x69bea3a1, 0x7c534292, 0x1dec5ee1 
};

__device__ void md5(char* message, int length, unsigned int* digest) 
{
	// Encryption init values (key)
   	unsigned int a0 = 0x67452301;
	unsigned int b0 = 0xefcdab89; 
   	unsigned int c0 = 0x98badcfe; 
   	unsigned int d0 = 0x10325476; 

	// Init values
	unsigned int A = a0;
	unsigned int B = b0;
	unsigned int C = c0;
	unsigned int D = d0;

	// Encrypted message
	unsigned int M[16] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0};

	// Move the input password to the temp buffer message
	memcpy(M,message,length);
	// Attach a termination character at the end of the input password inside the temp buffer
	((char*)M)[length]=0x80;
	// Add the length * 8 value in last but one position of the array
	M[14]=length*8;
	// Loop 64 times
	for (int i = 0;i < 64;i++) 
	{
		// Encryption logic
		unsigned int F = (B & C) | ((~B) & D);
		unsigned int G = (D & B) | ((~D) & C);
		unsigned int H = B ^ C ^ D;
		unsigned int I = C ^ (B | (~D));
		unsigned int tempD = D;
		D = C;
		C = B;
		unsigned int X = I;
		unsigned int g = (7*i) & 15;
		if (i < 48) { X = H; g = (3*i+5) & 15; }
		if (i < 32) { X = G; g = (5*i+1) & 15; }
		if (i < 16) { X = F; g = i; }

		unsigned int tmp = A + X + k_table[i] + M[g];
		B = B + ((tmp << s_table[i]) | ((tmp & 0xffffffff) >> (32-s_table[i])));
		A = tempD;
	}
	// Store the encrypted password
	digest[0] = a0 + A;
	digest[1] = b0 + B;
	digest[2] = c0 + C;
	digest[3] = d0 + D;
}

__device__ int pwd_num = 0;

__global__ void passwordCrackKernel(char* match_d)
{
    unsigned int dg[4];
    char pwd[PWD_LEN + 1];
    // Get the thread and block ids for iteration
    int i = blockIdx.x;
    int j = blockIdx.y;
    int k = blockIdx.z;
    int l = threadIdx.x;
    int m = threadIdx.y;
    int n = threadIdx.z;

    // Index validity check
    if (i + j + k + l + m + n <= PWD_LEN * (NUM_ALPHABETS - 1))
    {
        // Initialize the password with a character sequence for comparison
        pwd[0] = 'a' + i;
        pwd[1] = 'a' + j;
        pwd[2] = 'a' + k;
        pwd[3] = 'a' + l;
        pwd[4] = 'a' + m;
        pwd[5] = 'a' + n;
        pwd[6] = 0;

        // Get the encrypted version of the input password sequence
        md5(pwd, PWD_LEN, dg);
        // Loop through all the digest combination for a match
        for (int ii = 0;ii < MAX_DG ;ii++)
        {
            // Check for a encryption hit
            if (( dg[0] == digests_6letters[ii*4] ) && ( dg[1] == digests_6letters[ii*4+1] ) && ( dg[2] == digests_6letters[ii*4+2] ) && ( dg[3] == digests_6letters[ii*4+3] )) 
            {
                // Use a temporary pointer to point to next available empty location
                char* temp = &match_d[pwd_num];
                // Copy the matching password to the location specified
                memcpy(temp,pwd,PWD_LEN);
                // Increement the password indicator to the next location
                pwd_num += PWD_LEN;
            }
        }
    }
}

// Main function
int main()
{
    // Local variables for host and device
    char match_h[PWD_LEN*MAX_DG];
    char* match_d;

    // Allocate the memory for the pointers
    hipMalloc((void**) &match_d, PWD_LEN*MAX_DG);

    // CUDA kernel block and grid dimensions
    dim3 threadsPerBlock(NUM_ALPHABETS, NUM_ALPHABETS, NUM_ALPHABETS);
    dim3 blocksPerGrid(NUM_ALPHABETS, NUM_ALPHABETS, NUM_ALPHABETS);

	clock_t begin = clock();

    // Invoke cuda kernel
    passwordCrackKernel<<<blocksPerGrid, threadsPerBlock>>>(match_d);

    // Synchronize all the threads
    hipDeviceSynchronize();

    // Copy the result from device to host
    hipMemcpy(match_h, match_d, PWD_LEN*MAX_DG, hipMemcpyDeviceToHost);

    // Print the password
    for (int i = 0; i < PWD_LEN*MAX_DG; i = i+PWD_LEN)
    {
        printf("%c", match_h[i]);
        printf("%c", match_h[i+1]);
        printf("%c", match_h[i+2]);
        printf("%c", match_h[i+3]);
        printf("%c", match_h[i+4]);
        printf("%c\n", match_h[i+5]);
    }

	printf("\nElapsed time: %f seconds\n", (double)(clock() - begin) / CLOCKS_PER_SEC);

    // Free the pointers
    hipFree(match_d);
}